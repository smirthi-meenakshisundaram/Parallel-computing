#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image/stb_image_write.h"

void colorToGrayscale(unsigned char *gray_img, unsigned char * color_img, int width, int height);

const int IS_PNG = 1;
const int IS_JPG = 2;
const int DESIRED_CHANNELS = 3;
const int MAX_NAME_LENGTH = 500;
int main(int argc, char *argv[]) {
    if (argc < 4){
        printf("Usage: color2Grayscale ${input color image file} ${output grayscale image file} ${image type}\n Image Types:\n\t1: PGN\n\t2: JPG");
	exit(-1);
    }

    int width, height, channels, type;
    char  in_name[MAX_NAME_LENGTH], out_name[MAX_NAME_LENGTH];
    strcpy(in_name, argv[1]);
    strcpy(out_name, argv[2]);
    type = atoi(argv[3]);

 char *dev_color_img, *dev_gray_img, *dev_height,*dev_width; 

    unsigned char *color_img = stbi_load(in_name, &width, &height, &channels, 0); // load and conver the image to 3 channels (ignore the transparancy channel)
    if(color_img == NULL) {
        printf("Error in loading the image\n");
        exit(-1);
    }
    printf("Loaded image %s with a width of %dpx, a height of %dpx and %d channels\n", in_name, width, height, channels);

    // Convert the input image to gray
    int gray_channels = channels == 4 ? 2 : 1;
    size_t gray_img_size = width * height * gray_channels;
size_t color_img_size = width * height * channels;
    
    unsigned char *gray_img = (unsigned char *)malloc(gray_img_size);
    if(gray_img == NULL) {
        printf("Unable to allocate memory for the gray image.\n");
        exit(1);
    }
    printf("Create a image array with a width of %dpx, a height of %dpx and %d channels\n", width, height, gray_channels);

	 hipMalloc( (void**)&dev_color_img, (color_img_size* sizeof(unsigned char) );
     hipMalloc( (void**)&dev_gray_img, gray_img_size * sizeof(unsigned char) );
     hipMalloc( (void**)&dev_width, width* sizeof(int) );
	hipMalloc( (void**)&dev_height, height * sizeof(int));
     hipMemcpy( dev*color_img, color_img, color_img_size* sizeof(unsigned char), hipMemcpyHostToDevice );
     

int threadsperblock = 256;
int blockspergrid = (N + threadsperblock - 1) /threadsperblock;

   colorToGrayscale<<<blockspergrid,threadsperblock>>>(dev_gray_img, dev_color_img,dev_width,dev_height);

hipMemcpy( gray_img, dev_gray_img, gray_img_size * sizeof(unsigned 
char),hipMemcpyDeviceToHost );

    if (type == IS_PNG)
    	stbi_write_png(out_name, width, height, gray_channels, gray_img, width * gray_channels);

    else
        if (type == IS_JPG)
            stbi_write_jpg(out_name, width, height, gray_channels, gray_img, 100); //The last parameter of the stbi_write_jpg function is a quality parameter that goes from 1 to 100. Since JPG is a lossy image format, you can chose how much data is dropped at save time. Lower quality means smaller image size on disk and lower visual image quality.
    printf("Wrote image %s with a width of %dpx, a height of %dpx and %d channels\n", out_name, width, height, channels);

    stbi_image_free(gray_img); 

	free(color_img);
    	free(gray_img);
    	hipFree (dev_color_img);
    	hipFree (dev_gray_img);
    	hipFree (dev_height);
	hipFree (dev_width);
	
	return 0;
}


__global__ void colorToGrayscale(unsigned char *gray_img, unsigned char * color_img, int width, int height){


    unsigned char pixel[DESIRED_CHANNELS];


	int x = threadIdx.x + blockIdx.x * blockDim.x;
        int y = threadIdx.y + blockIdx.y * blockDim.x;

     
           //If the input image has a transparency channel this will be simply copied to the second channel of the gray image, while the first channel of the gray image will contain the gray pixel values. If the input image has three channels, the output image will have only one channel with the gray data.
 		
if(x<width && y<height)
{
           // int greyOffset = row * width + col;

		int greyOffset = y * width + x;
            int rgbOffset = greyOffset * DESIRED_CHANNELS;
            pixel[0] = color_img[rgbOffset];
            pixel[1] = color_img[rgbOffset + 1];
            pixel[2] = color_img[rgbOffset + 2];

            gray_img[greyOffset] = pixel[0] * 0.3 + pixel[1] * 0.58 + pixel[2] * 0.11;
        }
    }

